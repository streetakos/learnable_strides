// Copyright (c) OpenMMLab. All rights reserved
#include "stride_conv_cuda_kernel.cu"
#include "pytorch_cuda_helper.hpp"
#include <cstdio>
#include <cmath>

void stride_im2col(Tensor data_im,  const int channels,
                       const int height, const int width, const int ksize_h,
                       const int ksize_w, const float stride_h, const float stride_w ,const int pad_h, const int pad_w,
                       const int dilation_h, const int dilation_w,
                       const int parallel_imgs,
                       Tensor data_col) {
  // num_axes should be smaller than block size
  // todo: check parallel_imgs is correctly passed in


  int dH = (int)stride_h;
  int dW = (int)stride_w;

  int height_col = (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / dH + 1;
  int width_col = (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / dW + 1;

  int num_kernels = channels * height_col * width_col * parallel_imgs;


  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_im.scalar_type(), "stride_im2col_gpu", ([&] {
        const scalar_t *data_im_ = data_im.data_ptr<scalar_t>();
        scalar_t *data_col_ = data_col.data_ptr<scalar_t>();

        stride_im2col_gpu_kernel<<<GET_BLOCKS(num_kernels),
                                       THREADS_PER_BLOCK, 0,
                                       at::cuda::getCurrentCUDAStream()>>>(
            num_kernels, data_im_, height, width, ksize_h,  ksize_w,
            stride_h, stride_w, pad_h, pad_w, dilation_h, dilation_w,
            parallel_imgs, channels,
            height_col, width_col, data_col_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}

void stride_col2im_pytorch(Tensor data_col, const int channels,
                       const int height, const int width, const int ksize_h,
                       const int ksize_w,const float stride_h, const float stride_w, const int pad_h, const int pad_w,
                       const int dilation_h, const int dilation_w,
                       const int parallel_imgs, Tensor grad_im) {

   int dH = (int)stride_h;
   int dW = (int)stride_w;

  int num_kernels = channels * height * width;
  //Compute output height and width
  int height_col = (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / dH + 1;
  int width_col = (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / dW + 1;
  //int num_kernels = channels * ksize_h * ksize_w * height_col * width_col * parallel_imgs;
  //int channel_per_deformable_group = channels / deformable_group;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.scalar_type(), "stride_col2im_gpu", ([&] {
        const scalar_t *data_col_ = data_col.data_ptr<scalar_t>();
        scalar_t *grad_im_ = grad_im.data_ptr<scalar_t>();

        stride_col2im_pytorch_gpu_kernel<<<GET_BLOCKS(num_kernels),
                                       THREADS_PER_BLOCK, 0,
                                       at::cuda::getCurrentCUDAStream()>>>(
            num_kernels, data_col_, channels, height, width,
            ksize_h, ksize_w, stride_h, stride_w, pad_h, pad_w, dilation_h,
            dilation_w, parallel_imgs, height_col, width_col, grad_im_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}

void stride_col2im(Tensor data_col, const int channels,
                       const int height, const int width, const int ksize_h,
                       const int ksize_w,const float stride_h, const float stride_w, const int pad_h, const int pad_w,
                       const int dilation_h, const int dilation_w,
                       const int parallel_imgs, Tensor grad_im) {

  int dH = (int)stride_h;
  int dW = (int)stride_w;

  // todo: make sure parallel_imgs is passed in correctly
  int height_col = (height + 2 * pad_h - (dilation_h * (ksize_h - 1) + 1)) / dH + 1;
  int width_col = (width + 2 * pad_w - (dilation_w * (ksize_w - 1) + 1)) / dW + 1;
  int num_kernels = channels * ksize_h * ksize_w * height_col * width_col * parallel_imgs;
  //int channel_per_deformable_group = channels / deformable_group;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      data_col.scalar_type(), "stride_col2im_gpu", ([&] {
        const scalar_t *data_col_ = data_col.data_ptr<scalar_t>();
        scalar_t *grad_im_ = grad_im.data_ptr<scalar_t>();

        stride_col2im_gpu_kernel<<<GET_BLOCKS(num_kernels),
                                       THREADS_PER_BLOCK, 0,
                                       at::cuda::getCurrentCUDAStream()>>>(
            num_kernels, data_col_, channels, height, width,
            ksize_h, ksize_w, stride_h,stride_w,pad_h, pad_w, dilation_h,
            dilation_w, parallel_imgs, height_col, width_col, grad_im_);
      }));
  AT_CUDA_CHECK(hipGetLastError());
}


void stride_conv_shape_check(Tensor input, Tensor *gradOutput,
                             Tensor weight, int kH, int kW,float stride_h, float stride_w,
                             int padH, int padW, int dilationH, int dilationW,
                             int group) {

  TORCH_CHECK(
      weight.ndimension() == 4,
      "4D weight tensor (nOutputPlane,nInputPlane,kH,kW) expected, but got: %s",
      weight.ndimension());

  TORCH_CHECK(weight.is_contiguous(), "weight tensor has to be contiguous");

  TORCH_CHECK(kW > 0 && kH > 0,
              "kernel size should be greater than zero, but got kH: %d kW: %d",
              kH, kW);

  TORCH_CHECK((weight.size(2) == kH && weight.size(3) == kW),
              "kernel size should be consistent with weight, ",
              "but got kH: %d kW: %d weight.size(2): %d, weight.size(3): %d",
              kH, kW, weight.size(2), weight.size(3));

  float dfH = stride_h;
  float dfW = stride_w;

  TORCH_CHECK( dfH >= 1.0 && dfW >= 1.0,
              "stride should be greater equal than one, but got dH: %f dW: %f", dfH,
              dfW);

  TORCH_CHECK(
      dilationW > 0 && dilationH > 0,
      "dilation should be greater than 0, but got dilationH: %d dilationW: %d",
      dilationH, dilationW);

  int ndim = input.ndimension();
  int dimf = 0;
  int dimh = 1;
  int dimw = 2;

  if (ndim == 4) {
    dimf++;
    dimh++;
    dimw++;
  }

  TORCH_CHECK(ndim == 3 || ndim == 4,
              "3D or 4D input tensor expected but got: %s", ndim);

  int dH = (int)stride_h;
  int dW = (int)stride_w;

  long nInputPlane = weight.size(1) * group;
  long inputHeight = input.size(dimh);
  long inputWidth = input.size(dimw);
  long nOutputPlane = weight.size(0);
  long outputHeight =
      (inputHeight + 2 * padH - (dilationH * (kH - 1) + 1)) / dH + 1;
  long outputWidth =
      (inputWidth + 2 * padW - (dilationW * (kW - 1) + 1)) / dW + 1;




  if (outputWidth < 1 || outputHeight < 1)
    AT_ERROR(
        "Given input size: (%ld x %ld x %ld). "
        "Calculated output size: (%ld x %ld x %ld). Output size is too small",
        nInputPlane, inputHeight, inputWidth, nOutputPlane, outputHeight,
        outputWidth);

  TORCH_CHECK(input.size(1) == nInputPlane,
              "invalid number of input planes, expected: %d, but got: %d",
              nInputPlane, input.size(1));

  TORCH_CHECK((inputHeight >= kH && inputWidth >= kW),
              "input image is smaller than kernel");



  if (gradOutput != NULL) {
    TORCH_CHECK(
        gradOutput->size(dimf) == nOutputPlane,
        "invalid number of gradOutput planes, expected: %d, but got: %d",
        nOutputPlane, gradOutput->size(dimf));

    TORCH_CHECK(
        (gradOutput->size(dimh) == outputHeight &&
         gradOutput->size(dimw) == outputWidth),
        "invalid size of gradOutput, expected height: %d width: %d , but "
        "got height: %d width: %d",
        outputHeight, outputWidth, gradOutput->size(dimh),
        gradOutput->size(dimw));
  }
}

void StrideConvForwardCUDAKernelLauncher(Tensor input, Tensor weight,
                                         Tensor bias, Tensor output,
                                         Tensor columns, Tensor ones, int kW,
                                         int kH, float stride_h, float stride_w,
                                         int padW, int padH, int dilationW, int dilationH,
                                         int group, int im2col_step) {


  at::DeviceGuard guard(input.device());


  stride_conv_shape_check(input, NULL, weight, kH, kW,stride_h,stride_w,padH,
                          padW, dilationH, dilationW, group );

  int batch = 1;
  if (input.ndimension() == 3) {
    // Force batch
    batch = 0;
    input.unsqueeze_(0);
  }


  long batchSize = input.size(0);
  long nInputPlane = input.size(1);
  long inputHeight = input.size(2);
  long inputWidth = input.size(3);

  long nOutputPlane = weight.size(0);

  // Get the integer part of strides in order to compute the output
  int dH = (int)stride_h;
  int dW = (int)stride_w;

  long outputWidth =
      (inputWidth + 2 * padW - (dilationW * (kW - 1) + 1)) / dW + 1;
  long outputHeight =
      (inputHeight + 2 * padH - (dilationH * (kH - 1) + 1)) / dH + 1;


  output = output.view({batchSize / im2col_step, im2col_step, nOutputPlane,
                        outputHeight, outputWidth});

  columns = at::zeros(
      {nInputPlane * kW * kH, im2col_step * outputHeight * outputWidth},
      input.options());

  if (ones.ndimension() != 2 ||
      ones.size(0) * ones.size(1) < outputHeight * outputWidth) {
    ones = at::ones({outputHeight, outputWidth}, input.options());
  }

  input = input.view({batchSize / im2col_step, im2col_step, nInputPlane,
                      inputHeight, inputWidth});


  Tensor output_buffer = at::zeros({batchSize / im2col_step, nOutputPlane,
                                    im2col_step * outputHeight, outputWidth},
                                   output.options());

  output_buffer = output_buffer.view(
      {output_buffer.size(0), group, output_buffer.size(1) / group,
       output_buffer.size(2), output_buffer.size(3)});


  Tensor bias_g = bias.view({group, nOutputPlane/group});

  for (int elt = 0; elt < batchSize / im2col_step; elt++) {
    stride_im2col(input[elt], nInputPlane, inputHeight,
                      inputWidth, kH, kW,   stride_h,  stride_w, padH, padW, dilationH,
                      dilationW, im2col_step, columns);

    columns = columns.view({group, columns.size(0) / group, columns.size(1)});
    weight = weight.view({group, weight.size(0) / group, weight.size(1),
                          weight.size(2), weight.size(3)});

    for (int g = 0; g < group; g++) {

	  output_buffer[elt][g]  = (output_buffer[elt][g]
                                  .flatten(1).addmm_(weight[g].flatten(1), columns[g]) + bias_g[g].view({bias_g[g].size(0),1})  ).view_as(output_buffer[elt][g]) ;
	  }
    columns = columns.view({columns.size(0) * columns.size(1), columns.size(2)});
    weight = weight.view({weight.size(0) * weight.size(1), weight.size(2),
                          weight.size(3), weight.size(4)});
  }

  output_buffer = output_buffer.view(
      {output_buffer.size(0), output_buffer.size(1) * output_buffer.size(2),
       output_buffer.size(3), output_buffer.size(4)});

  output_buffer = output_buffer.view({batchSize / im2col_step, nOutputPlane,
                                      im2col_step, outputHeight, outputWidth});
  output_buffer.transpose_(1, 2);
  output.copy_(output_buffer);
  output = output.view({batchSize, nOutputPlane, outputHeight, outputWidth});

  input = input.view({batchSize, nInputPlane, inputHeight, inputWidth});

  if (batch == 0) {
    output = output.view({nOutputPlane, outputHeight, outputWidth});
    input = input.view({nInputPlane, inputHeight, inputWidth});
  }

}

void StrideConvBackwardInputCUDAKernelLauncher(
    Tensor input, Tensor gradOutput, Tensor gradInput,
    Tensor gradBias, Tensor weight, Tensor columns, int kW, int kH, float stride_h, float stride_w,
	int padW, int padH, int dilationW, int dilationH, int group,
    int im2col_step) {

  at::DeviceGuard guard(input.device());


  int batch = 1;

  if (input.ndimension() == 3) {
    // Force batch
    batch = 0;
    input = input.view({1, input.size(0), input.size(1), input.size(2)});
    gradOutput = gradOutput.view({1, gradOutput.size(0), gradOutput.size(1), gradOutput.size(2)});
  }

  long batchSize = input.size(0);
  long nInputPlane = input.size(1);
  long inputHeight = input.size(2);
  long inputWidth = input.size(3);

  long nOutputPlane = weight.size(0);

  long ws0 = 	weight.size(0);
  long ws1 = 	weight.size(1);
  long ws2 = 	weight.size(2);
  long ws3 = 	weight.size(3);

  int dH = (int)stride_h;
  int dW = (int)stride_w;

  long outputWidth = (inputWidth + 2 * padW - (dilationW * (kW - 1) + 1)) / dW + 1;
  long outputHeight = (inputHeight + 2 * padH - (dilationH * (kH - 1) + 1)) / dH + 1;

  //TORCH_CHECK((offset.size(0) == batchSize), 3, "invalid batch size of offset");
  gradInput = gradInput.view({batchSize, nInputPlane, inputHeight, inputWidth});
  columns = at::zeros({nInputPlane * kW * kH, im2col_step * outputHeight * outputWidth},
      input.options());

  // change order of grad output
  gradOutput = gradOutput.view({batchSize / im2col_step, im2col_step,  nOutputPlane, outputHeight, outputWidth});
  gradOutput.transpose_(1, 2);
  gradInput = gradInput.view({batchSize / im2col_step, im2col_step, nInputPlane, inputHeight, inputWidth});
  input = input.view({batchSize / im2col_step, im2col_step, nInputPlane, inputHeight, inputWidth});

  gradBias = gradBias.view({group, nOutputPlane/group});

  for (int elt = 0; elt < batchSize / im2col_step; elt++) {
	// Multiply each weight with each outputpixel and sum the output dimentions via multiplication
    // divide into groups
    columns = columns.view({group, columns.size(0) / group, columns.size(1)});
    weight = weight.view({ group, weight.size(0) / group, weight.size(1), weight.size(2), weight.size(3)});
    gradOutput = gradOutput.view({ gradOutput.size(0), group, gradOutput.size(1) / group, gradOutput.size(2), gradOutput.size(3), gradOutput.size(4)});

    // From the chain rule:
	   //G_offset = gradOutput * W * G_interpolated_value,  Compute: gradOutput * W
    for (int g = 0; g < group; g++) {
      columns[g] = columns[g].addmm_(weight[g].flatten(1).transpose(0, 1),
                                     gradOutput[elt][g].flatten(1), 0.0f, 1.0f);
      gradBias[g] = gradOutput[elt][g].transpose(0, 1).sum( {0,2,3}, false) ; ///////////////////////////////////////////////////////////////////
    }

    columns = columns.view({columns.size(0) * columns.size(1), columns.size(2)});
    gradOutput = gradOutput.view({gradOutput.size(0), gradOutput.size(1) * gradOutput.size(2), gradOutput.size(3), gradOutput.size(4), gradOutput.size(5)});

    weight = weight.view({weight.size(0) * weight.size(1), weight.size(2),
                          weight.size(3), weight.size(4)});

  }



  gradOutput.transpose_(1, 2);
  gradOutput = gradOutput.view({batchSize, nOutputPlane, outputHeight, outputWidth});

  ////////////////////////////////////////////////////////////////////////////////////////////////////////////
  // Code adopted from pytorch for avoiding atomicadd operation (Deterministic)
  ///*
  gradInput = gradInput.view({batchSize , nInputPlane, inputHeight, inputWidth});
  weight = weight.view({  ws0 , ws1, ws2, ws3});

  for (int elt = 0; elt < batchSize ; elt++) {
	Tensor columnsa = at::zeros({nInputPlane * kW * kH, outputHeight * outputWidth},input.options());

	columnsa = columnsa.addmm_(weight.flatten(1).transpose(0, 1),
                                     gradOutput[elt].flatten(1), 0.0f, 1.0f);

	stride_col2im_pytorch(columnsa, nInputPlane, inputHeight,
                      inputWidth, kH, kW, stride_h, stride_w, padH, padW, dilationH,
                      dilationW, im2col_step, gradInput[elt]);
  }
  ////////////////////////////////////////////////////////////////////////////////////////////////////////////


  gradInput = gradInput.view({batchSize, nInputPlane, inputHeight, inputWidth});
  input = input.view({batchSize, nInputPlane, inputHeight, inputWidth});
  gradBias = gradBias.view({nOutputPlane});


  if (batch == 0) {
    gradOutput = gradOutput.view({nOutputPlane, outputHeight, outputWidth});
    input = input.view({nInputPlane, inputHeight, inputWidth});
    gradInput = gradInput.view({nInputPlane, inputHeight, inputWidth});
  }

}

void StrideConvBackwardParametersCUDAKernelLauncher(
    Tensor input, Tensor gradOutput, Tensor gradWeight,
    Tensor columns, Tensor ones, int kW, int kH, float stride_h, float stride_w,
    int padW, int padH, int dilationW, int dilationH, int group,
    float scale, int im2col_step) {

  at::DeviceGuard guard(input.device());

  int batch = 1;

  if (input.ndimension() == 3) {
    // Force batch
    batch = 0;
    input = input.view( at::IntList({1, input.size(0), input.size(1), input.size(2)}));
    gradOutput = gradOutput.view({1, gradOutput.size(0), gradOutput.size(1), gradOutput.size(2)});
  }

  long batchSize = input.size(0);
  long nInputPlane = input.size(1);
  long inputHeight = input.size(2);
  long inputWidth = input.size(3);

  long nOutputPlane = gradWeight.size(0);

  int dH = (int)stride_h;
  int dW = (int)stride_w;

  long outputWidth =  (inputWidth + 2 * padW - (dilationW * (kW - 1) + 1)) / dW + 1;
  long outputHeight = (inputHeight + 2 * padH - (dilationH * (kH - 1) + 1)) / dH + 1;

  //TORCH_CHECK((offset.size(0) == batchSize), "invalid batch size of offset");

  columns = at::zeros({nInputPlane * kW * kH, im2col_step * outputHeight * outputWidth}, input.options());

  gradOutput = gradOutput.view({batchSize / im2col_step, im2col_step, nOutputPlane, outputHeight, outputWidth});
  gradOutput.transpose_(1, 2);

  Tensor gradOutputBuffer = at::zeros_like(gradOutput);
  gradOutputBuffer = gradOutputBuffer.view({batchSize / im2col_step, nOutputPlane, im2col_step, outputHeight, outputWidth});
  gradOutputBuffer = gradOutputBuffer.contiguous();
  gradOutputBuffer.copy_(gradOutput);
  gradOutputBuffer = gradOutputBuffer.view({batchSize / im2col_step, nOutputPlane, im2col_step * outputHeight, outputWidth});

  gradOutput.transpose_(1, 2);
  gradOutput =  gradOutput.view({batchSize, nOutputPlane, outputHeight, outputWidth});

  input = input.view({batchSize / im2col_step, im2col_step, nInputPlane, inputHeight, inputWidth});

  for (int elt = 0; elt < batchSize / im2col_step; elt++) {
    stride_im2col(input[elt], nInputPlane, inputHeight,
                      inputWidth, kH, kW, stride_h, stride_w, padH, padW, dilationH,
                      dilationW, im2col_step, columns);

    // divide into group
    gradOutputBuffer = gradOutputBuffer.view( {gradOutputBuffer.size(0), group, gradOutputBuffer.size(1) / group, gradOutputBuffer.size(2), gradOutputBuffer.size(3)});
    columns = columns.view({group, columns.size(0) / group, columns.size(1)});
    gradWeight = gradWeight.view({group, gradWeight.size(0) / group, gradWeight.size(1), gradWeight.size(2), gradWeight.size(3)});

    for (int g = 0; g < group; g++) {
      gradWeight[g] = gradWeight[g].flatten(1).addmm_(gradOutputBuffer[elt][g].flatten(1),
                                                     columns[g].transpose(1, 0), 1.0, scale).view_as(gradWeight[g]);
    }
    gradOutputBuffer = gradOutputBuffer.view( {gradOutputBuffer.size(0), gradOutputBuffer.size(1) * gradOutputBuffer.size(2), gradOutputBuffer.size(3), gradOutputBuffer.size(4)});
    columns = columns.view({columns.size(0) * columns.size(1), columns.size(2)});
    gradWeight = gradWeight.view({gradWeight.size(0) * gradWeight.size(1), gradWeight.size(2), gradWeight.size(3), gradWeight.size(4)});
  }

  input = input.view({batchSize, nInputPlane, inputHeight, inputWidth});
  //offset = offset.view( {batchSize, deformable_group * 2 * kH * kW, outputHeight, outputWidth});

  if (batch == 0) {
    gradOutput = gradOutput.view({nOutputPlane, outputHeight, outputWidth});
    input = input.view({nInputPlane, inputHeight, inputWidth});
  }

}
